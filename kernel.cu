/*
	kernel.cu
	Holds the kernel for the main program
*/


#include <hip/hip_runtime.h>
#include <iostream>

#define BLOCK_WIDTH 32
#define cuda_check_errors(val) check( (val), #val, __FILE__, __LINE__)

using namespace std;

template<typename T>
void check(T err, const char* const func, const char* const file, 
			const int line) {
	if (err != hipSuccess) {
		cerr << "CUDA error at: " << file << ":" << line << endl;
		cerr << hipGetErrorString(err) << " " << func << endl;
		exit(1);
	}
}

__global__
void rgba_to_grey(uchar4 *const d_rgba, unsigned char *const d_grey, 
					size_t rows, size_t cols) {
	size_t j = blockIdx.y * blockDim.y + threadIdx.y;
	size_t i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= rows || j >= cols) 
		return;
	
	uchar4 p = d_rgba[i * cols + j];
	d_grey[i * cols + j] = (unsigned char) (0.299f * p.x + 0.587f * p.y + 0.114f * p.z);
}

void rgba_to_grey_launcher(uchar4 *const d_rgba, unsigned char *const d_grey,
							size_t rows, size_t cols) {
    const dim3 block_size (BLOCK_WIDTH, BLOCK_WIDTH, 1);
    unsigned int grid_x = (unsigned int) (rows / BLOCK_WIDTH + 1);
    unsigned int grid_y = (unsigned int) (cols / BLOCK_WIDTH + 1);
    const dim3 grid_size (grid_x, grid_y, 1);
    rgba_to_grey<<<grid_size, block_size>>>(d_rgba, d_grey, rows, cols);
    hipDeviceSynchronize();
    cuda_check_errors(hipGetLastError());
}
